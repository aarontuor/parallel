#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void printmatrix(float * mat, int n){
    int i, j;
    printf("\n");
    for (i = 0; i < n; i++){
        for (j = 0; j < n; j++){
            printf("%f ", mat[i*n + j]);
        }
        printf("\n");
    }
}

void writematrix(FILE *outfile, float * mat, int n){
    int i, j;
    for (i = 0; i < n; i++){
        for (j = 0; j < n; j++){
            fprintf(outfile, "%f ", mat[i*n + j]);
        }
        fprintf(outfile, "\n");
    }
}

__global__ void jacobi_iter(float *dmatrix, float *dmatrix2, float *fix_points, int n){
    int x = blockIdx.x;
    int y = threadIdx.x;
    if (fix_points[x*n + y] != 1.0){
        dmatrix2[x*n + y] = dmatrix[x*n + 1 + y]*0.25;
        dmatrix2[x*n + y] += dmatrix[x*n - 1 + y]*0.25;
        dmatrix2[x*n + y] += dmatrix[(x-1)*n + y]*0.25;
        dmatrix2[x*n + y] += dmatrix[(x+1)*n + y]*0.25;
    }
}

__global__ void max_diff_reduce(float *mat1, float *mat2, float *reduced_mat) {
    extern __shared__ float sdata[];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = mat1[i] - mat2[i];
	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) {
			sdata[tid] = fmax((float)sdata[tid], (float)sdata[tid + s]);
		}
		__syncthreads();
	}
	// write result for this block to global mem
	if (tid == 0) {
		reduced_mat[blockIdx.x] = sdata[0];
	}
}



int main(int argc, char **argv){

    char * filename;
    filename =  argv[1];
    FILE *spec = fopen(filename, "r");
    FILE *outfile = fopen(argv[2], "w");
    int n, i, j, iter; //size of square matrix
    float delta; //for ending iterations
    int check;
    check = fscanf(spec, "%d %f\n", &n, &delta);
    int msize = n*n*sizeof(float);


    float *matrix = (float *)malloc(msize); //host matrix
    float *matrix2 = (float *)malloc(msize); //host matrix
    float *fix_points = (float *)malloc(msize); //host matrix
    for (i = 0; i < n*n; i++){
         fix_points[i] = 0.0;
    }
	float *reduced_mat = (float *)malloc(n*sizeof(float));
    float *dmatrix; // kernel matrix for alternating
    hipMalloc((void**) &dmatrix, msize);
    float *dmatrix2; // kernel matrix for alternating
    hipMalloc((void**) &dmatrix2, msize);
    float *dfix_points; // kernel matrix fixed points
    hipMalloc((void**) &dfix_points, msize);
	float *dreduced_mat;
	hipMalloc((void**) &dreduced_mat, n*sizeof(float));

	int urow, ucol, drow, dcol;
    float val;

    while (fscanf(spec, "\n%f %d %d %d %d\n", &val, &urow, &ucol, &drow, &dcol) == 5){
        printf("%f %d %d %d %d\n", val, urow, ucol, drow, dcol);
        for (i=urow; i<=drow; i++){
            for(j=ucol; j<=dcol; j++){
                matrix[i*n + j] = val;
                fix_points[i*n + j] = 1.0;
            }
        }
    }
    //printmatrix(matrix, n);
    fclose(spec);
    //printf("msize: %d delta %f\n", n*n, delta);
    hipMemcpy(dmatrix, matrix, msize, hipMemcpyHostToDevice);
    hipMemcpy(dmatrix2, matrix, msize, hipMemcpyHostToDevice);
    hipMemcpy(dfix_points, fix_points, msize, hipMemcpyHostToDevice);
    float maxdiff = 2.0*delta;
    dim3 dimGrid(n,1);
    dim3 dimBlock(n,1,1);
    iter = 0;
    maxdiff = 3;
    while(maxdiff > delta){

        iter += 1;
        jacobi_iter<<<dimGrid, dimBlock>>>(dmatrix, dmatrix2, dfix_points, n);
        hipDeviceSynchronize();
        hipMemcpy(matrix, dmatrix2, msize, hipMemcpyDeviceToHost);
        jacobi_iter<<<dimGrid, dimBlock>>>(dmatrix2, dmatrix, dfix_points, n);
        hipDeviceSynchronize();
        hipMemcpy(matrix2, dmatrix, msize, hipMemcpyDeviceToHost);
        //max_diff_reduce<<<dimGrid, dimBlock>>>(dmatrix2, dmatrix, dreduced_mat);
		// hipMemcpy(reduced_mat, dreduced_mat, n*sizeof(float), hipMemcpyDeviceToHost);
        maxdiff = fabs(matrix[0] - matrix2[0]);
        //maxdiff = reduced_mat[0];
		for(i=1; i<n*n; i++){
		 	maxdiff = fmax((float) maxdiff, (float) fabs((matrix[i]-matrix2[i])));
		}
        printf("\niter: %d maxdiff: %f delta: %f\n", iter, maxdiff, delta);
	}
    hipMemcpy(matrix, dmatrix2, msize, hipMemcpyDeviceToHost);
    //printmatrix(matrix, n);
    hipMemcpy(matrix, dmatrix, msize, hipMemcpyDeviceToHost);
    //printmatrix(matrix, n);
    writematrix(outfile, matrix2, n);
    fclose(outfile);
}
